

#include <hip/hip_runtime.h>
#include <math.h>

////////////////////////////////////////
// idx_gpu()
//
int idx_gpu(int r, int i2, int i1, int nr, int mt)
{
  int   idx;
  int   rbase;
  int   i2base;
  int   i1base;
  idx=0;

  rbase = r  * (mt+1)*(mt+1);
  i2base = i2 * (mt+1);
  i1base = i1;
  
  idx = rbase + i2base + i1base;

   return idx;
}

////////////////////////////////////////
// getNearestDataValue2_gpu()
//
double getNearestDataValue2_gpu(int _ndpth, double _minR, double _maxR, int _nlat, int _nlng, 
				double *_xn, double *_yn, double *_zn,
				double *_x,  double *_y , double *_z , double *_V,
				int index)
{
    double rad,dR,dataR;
    double d2;
    double xd,yd,zd;
    double tmpd2;
    double dataV;
    double localVeryLarge;
    int nr, di, ir; 
    
    double gx,gy,gz;
    gx = _xn[index];
    gy = _yn[index];
    gz = _zn[index];

    localVeryLarge=1E+99;

    rad=sqrt(gx*gx + gy*gy + gz*gz);

    nr = 0;
    dR = localVeryLarge;
    dataR = 0.;
    for ( ir=0 ; ir<_ndpth ; ir++ ){
      dataR = _minR + ir*(_maxR - _minR)/_ndpth;
      if (fabs(dataR - rad) < dR) {
	dR = fabs(dataR - rad) ;
	nr = _ndpth - ir;
      }
    }

    dataV = 0.;

    d2 = 1.E+99;
    
    for ( di=nr*_nlat*_nlng; di<nr*_nlat*_nlng + _nlat*_nlng; di++ ){
      xd=_x[di] - _xn[index];
      yd=_y[di] - _yn[index];
      zd=_z[di] - _zn[index];
      tmpd2=xd*xd + yd*yd + zd*zd;
      if ( tmpd2 < d2 ) {
	d2 = tmpd2;
	dataV = _V[di];
      }
    }

    return dataV;
}


////////////////////////////////////////
// importData_c_gpu()
//
extern "C" bool importData_c_gpu(int nr, int mt, 
		      int _ndpth, double _minR, double _maxR, int _nlat, int _nlng, 
		      double *_xn, double *_yn, double *_zn, double *_Vn,
		      double *_x,  double *_y , double *_z , double *_V)
{
  int ri,i2,i1,index;
  
  for ( ri=0 ; ri < nr ; ri++ ){
	for ( i2 = 0 ; i2 < mt+1 ; i2++ ){
	    for ( i1 = 0 ; i1 < mt+1 ; i1++ ) {
	      index=idx_gpu(ri,i2,i1,nr,mt);
	      _Vn[index] = getNearestDataValue2_gpu(_ndpth, _minR, _maxR, 
						    _nlat, _nlng,
						    _xn, _yn, _zn,
						    _x,  _y,  _z, _V,
						    index);
	    }
	}
    }

  return 0;
}



